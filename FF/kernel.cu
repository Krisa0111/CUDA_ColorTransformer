#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <hip/hip_runtime.h>
#include <>


__global__ void blackAndWhite(unsigned char* image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int index = (y * width + x) * 3;
        unsigned char r = image[index];
        unsigned char g = image[index + 1];
        unsigned char b = image[index + 2];
        unsigned char gray = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
        image[index] = gray;
        image[index + 1] = gray;
        image[index + 2] = gray;
    }
}

int main() {
    int width, height, channels;
    unsigned char* image = stbi_load("image.jpg", &width, &height, &channels, 0);

    unsigned char* d_image;
    hipMalloc(&d_image, width * height * channels);
    hipMemcpy(d_image, image, width * height * channels, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    blackAndWhite << <numBlocks, threadsPerBlock >> > (d_image, width, height);

    hipMemcpy(image, d_image, width * height * channels, hipMemcpyDeviceToHost);
    stbi_write_jpg("output.jpg", width, height, channels, image, 100);

    stbi_image_free(image);
    hipFree(d_image);
    return 0;
}